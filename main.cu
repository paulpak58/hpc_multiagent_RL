#include <iostream>
#include <hip/hip_runtime.h>
#include <random>
#include "random.cuh"
#include "alpha_beta.cuh"
using namespace std;

int main(int argc,char *argv[]){
	int n = atoi(argv[1]);
	int threads_per_block = atoi(argv[2]);

	// Timing Initialization
	hipEvent_t startEvent,stopEvent;
	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);
	float elapsedTime;

	// Random Initialization
	random_device entropy_source;
	mt19937_64 generator(entropy_source());
	uniform_real_distribution<float> dist(-1.0,1.0);

	// Create arrays in managed memory
	float *A,*B;
	hipMallocManaged(&A,n*sizeof(float));
	hipMallocManaged(&B,n*sizeof(float));
	for(int i=0;i<n;++i)
		A[i] = dist(generator);

	// Start time and call scan to conduct an inclusive scan
	hipEventRecord(startEvent,0);

	// ALPHA BETA ALGO

	hipEventRecord(stopEvent,0);
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&elapsedTime,startEvent,stopEvent);

 	printf("%f\n",B[n-1]);
	printf("%f\n",elapsedTime);

	// Clean up and free memory
	hipFree(A);
	hipFree(B);
	hipEventDestroy(startEvent);
	hipEventDestroy(stopEvent);
	return 0;
}
